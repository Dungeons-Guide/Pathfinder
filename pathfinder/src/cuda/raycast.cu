#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../raycast.h"
#include "PathfindRequest.h"


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__device__ int listIdx;

// Fix pixel perfect etherwarps and it would be fine.
__device__ bool ashadowcast(bool* req,
                          int lenX, int lenY, int lenZ,
                          float locX, float locY, float locZ,
                          float targetX, float  targetY, float  targetZ) {
    float dx = locX - targetX;
    float dy = locY - targetY;
    float dz = locZ - targetZ;
    short maxVal = max(abs(dx), max(abs(dy), abs(dz))) * 9;

    dx /= maxVal;
    dy /= maxVal;
    dz /= maxVal;

    for (short i = 0; i < maxVal-2; i++) {
        targetX += dx;
        targetY += dy;
        targetZ += dz;

        short x = (short) targetX;
        short y = (short) targetY;
        short z = (short) targetZ;
        int idx = z * lenX * lenY + y * lenX + x;
        if (req[idx]) {
            return false;
        }
    }
    return true;
}

__global__ void shadowcast(bool *req, int lenX, int lenY, int lenZ,
                           int offX, int offY, int offZ,
                           int toX, int toY, int toZ,
                           short targetX, short targetY, short targetZ,
                           float offset, int rad,
                           Coordinate* arr,
                           Coordinate* potentialShadowcasts, int cnt) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx > cnt) return;

    Coordinate location = potentialShadowcasts[idx];
    short locX = location.x;
    short locY = location.y;
    short locZ = location.z;

    short dx = targetX - locX;
    short dy = targetY - locY;
    short dz = targetZ - locZ;

    if (locX < offX || locY < offY || locZ < offZ || locX >= toX + offX || locY >= toY + offY || locZ >= toZ + offZ) {
        return;
    }
    if (dx * dx + dy * dy + dz * dz > rad * rad) {
        return;
    }


    for (int z = 0; z < 2; z++) {
        for (int x = 0; x < 2; x++) {
            for (int y = 0; y < 2; y++) {
                int idx = (locZ-z) * lenX * lenY + (locY-2-y) * lenX + locX-x;
                if (idx < 0) continue;
                if (req[idx]) {
                    goto label;
                }
            }
        }
    }
    return;
    label:

    for (int x = 0; x < 2; x++) {
        for (int z = 0; z < 2; z++) {
            for (int y = 0; y < 2; y++) {
                bool flag = false;
                int xTarget;
//                long begin = clock64();
                if (dx < 0) {
                    xTarget = targetX + 0.499999 + offset;
                } else {
                    xTarget = targetX + 0.500001 - offset;
                }
                flag |= ashadowcast(req, lenX, lenY, lenZ, locX + x/2.0, locY+ y / 2.0, locZ + z/2.0,
                                    xTarget, targetY + 0.499798, targetZ + 0.499889);
//                long end = clock64();
//                ahhh[atomicAdd(&listIdx2, 1)] = end-begin;

//                begin = clock64();
//                if (!flag) {
                    if (dy < 0) {
                        xTarget = targetY + 0.499798 + offset;
                    } else {
                        xTarget = targetY + 0.500202 - offset;
                    }
                    flag |= ashadowcast(req, lenX, lenY, lenZ, locX + x/2.0, locY + y / 2.0, locZ + z/2.0,
                                        targetX + 0.499999, xTarget, targetZ + 0.499889);
//                }
//                end = clock64();
//                ahhh[atomicAdd(&listIdx2, 1)] = end-begin;

//                begin = clock64();
//                if (!flag) {
                    if (dz < 0) {
                        xTarget = targetZ + 0.499889 + offset;
                    } else {
                        xTarget = targetZ + 0.500114 - offset;
                    }
                    flag |= ashadowcast(req, lenX, lenY, lenZ, locX + x/2.0, locY + y / 2.0, locZ+ z/2.0,
                                        targetX + 0.499999, targetY + 0.499798, xTarget);
//                }
//                end = clock64();
//                ahhh[atomicAdd(&listIdx2, 1)] = end-begin;


                if (flag) {
                    int val = atomicAdd(&listIdx, 1);
                    arr[val] = {
                            locX * 2 + x, locY * 2 + y, locZ * 2 + z
                    };
                }
            }

        }
    }
}

__global__ void findPotentialShadowcastDestinations(bool *req, int lenX, int lenY, int lenZ,
                                                    int fromY, int toY,
                                                    Coordinate* arr) {
    short locX = blockIdx.x * blockDim.x + threadIdx.x ;
    short locY = blockIdx.y * blockDim.y + threadIdx.y + fromY;
    short locZ = blockIdx.z * blockDim.z + threadIdx.z ;


    if (locX < 0 || locY < 0 || locZ < 0 || locX >= lenX || locY >= toY || locZ >= lenZ) {
        return;
    }

    for (int z = 0; z < 2; z++) {
        for (int x = 0; x < 2; x++) {
            for (int y = 0; y < 2; y++) {
                int idx = (locZ-z) * lenX * lenY + (locY-2-y) * lenX + locX-x;
                if (locZ - z < 0) continue;
                if (locY - y  -2 <0) continue;
                if (locX - x < 0) continue;

                if (req[idx]) {
                    goto label;
                }
            }
        }
    }
    return;
    label:

    for (int z = 0; z < 2; z++) {
        for (int x = 0; x < 2; x++) {
            for (int y = 0; y < 2; y++) {
                int idx = (locZ-z) * lenX * lenY + (locY-y) * lenX + locX-x;
                if (locZ - z < 0) continue;
                if (locY - y < 0) continue;
                if (locX - x < 0) continue;

                if (!req[idx]) {
                    goto label2;
                }
            }
        }
    }
    return;
    label2:

    int val = atomicAdd(&listIdx, 1);
    arr[val] = {
            locX, locY, locZ
    };
}

Coordinate* gpu_coord;
Coordinate* potentialShadowcasts;
int potentialShadowcastCount;
void setupCudaMemory() {
    gpuErrchk( hipMalloc((void**) &gpu_coord, sizeof(Coordinate) * GPU_RETURN_SIZE) );
    gpuErrchk( hipMalloc((void**) &potentialShadowcasts, sizeof(Coordinate) * 4000000) );
}

int setupPotentialShadowcasts(bool *req, int lenX, int lenY, int lenZ, int fromY, int toY) {
    int count = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(listIdx), &count, sizeof(int), 0, hipMemcpyHostToDevice);
    std::cout << lenX << "/"<<lenY << "/"<<lenZ<<"/"<<fromY<<"/"<<toY<<std::endl;

    dim3 blockDim(16,4,16);
    dim3 gridDim(ceil(lenX/16.0), ceil((toY-fromY)/4.0), ceil(lenZ/16.0));
    findPotentialShadowcastDestinations<<<gridDim, blockDim>>>(req, lenX, lenY, lenZ, fromY, toY,  potentialShadowcasts);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    gpuErrchk(hipMemcpyFromSymbol(&count, HIP_SYMBOL(listIdx), sizeof(int), 0, hipMemcpyDeviceToHost));

    std::cout << count << std::endl;
    potentialShadowcastCount = count;

    return count;
}



int callShadowCast(bool *req, int lenX, int lenY, int lenZ,
                    int fromX, int fromY, int fromZ, int toX, int toY, int toZ,
                   short targetX, short targetY, short targetZ, float offset, int rad, Coordinate* arr) {

    int count = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(listIdx), &count, sizeof(int), 0, hipMemcpyHostToDevice);;


    shadowcast<<<ceil(potentialShadowcastCount / 1024.0), 1024>>>
    (req, lenX, lenY, lenZ, fromX, fromY, fromZ, toX, toY, toZ,
                                      targetX, targetY, targetZ, offset, rad, gpu_coord, potentialShadowcasts,potentialShadowcastCount);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    gpuErrchk(hipMemcpyFromSymbol(&count, HIP_SYMBOL(listIdx), sizeof(int), 0, hipMemcpyDeviceToHost));

    gpuErrchk( hipMemcpy(arr, gpu_coord, sizeof(Coordinate ) * count, hipMemcpyDeviceToHost) );

    return count;
}
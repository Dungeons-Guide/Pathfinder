#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../raycast.h"
#include "PathfindRequest.h"


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__device__ int listIdx;

__global__ void shadowcast(bool *req, int lenX, int lenY, int lenZ,
                           int offX, int offY, int offZ,
                           int resLenx, int resLeny, int resLenz, short targetX, short targetY, short targetZ, int rad,
                           Coordinate* arr) {
    short locX = blockIdx.x * blockDim.x + threadIdx.x + offX;
    short locY = blockIdx.y * blockDim.y + threadIdx.y + offY;
    short locZ = blockIdx.z * blockDim.z + threadIdx.z + offZ;

//    x,y,z;
    short dx = targetX - locX;
    short dy = targetY - locY;
    short dz = targetZ - locZ;
    short maxVal = max(abs(dx), max(abs(dy), abs(dz)));


    float stepX = dx / (float) maxVal;
    float stepY = dy / (float) maxVal;
    float stepZ = dz / (float) maxVal;

    float currX = locX;
    float currY = locY;
    float currZ = locZ;

    if (locX < offX || locY < offY || locZ < offZ || locX >= resLenx + offX || locY >= resLeny + offY || locZ >= resLenz + offZ) {
        return;
    }
    if (dx * dx + dy * dy + dz * dz > rad * rad) {
        return;
    }

    for (short i = 0; i <= maxVal; i++) {
        currX += stepX;
        currY += stepY;
        currZ += stepZ;

        short x = (short) currX;
        short y = (short) currY;
        short z = (short) currZ;

        if (x < 0 || y < 0 || z < 0 || x >= lenX || y >= lenY || z >= lenZ) {
            return;
        }

        int idx = z * lenX * lenY + y * lenX + x;
        if (req[idx]) {
            return;
        }
    }
    int val = atomicAdd(&listIdx, 1);
    arr[val] = {
            locX,locY,locZ
    };
}

Coordinate* gpu_coord;
void setupCudaMemory() {
    gpuErrchk( hipMalloc((void**) &gpu_coord, sizeof(Coordinate) * GPU_RETURN_SIZE) );
}

int callShadowCast(bool *req, int lenX, int lenY, int lenZ,
                    int fromX, int fromY, int fromZ, int toX, int toY, int toZ,
                    short targetX, short targetY, short targetZ, int rad, Coordinate* arr) {

    int count = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(listIdx), &count, sizeof(int), 0, hipMemcpyHostToDevice);


    dim3 blockDim(10,10,10);
    dim3 gridDim(ceil((toX - fromX) / 10.0),ceil((toY - fromY) / 10.0),ceil((toZ - fromZ) / 10.0));
    shadowcast<<<gridDim, blockDim>>>(req, lenX, lenY, lenZ, fromX, fromY, fromZ,
                                      (toX - fromX), (toY - fromY), (toZ - fromZ),
                                      targetX, targetY, targetZ, rad, gpu_coord);

//    gpuErrchk( hipPeekAtLastError() );
//    gpuErrchk( hipDeviceSynchronize() );

    gpuErrchk(hipMemcpyFromSymbol(&count, HIP_SYMBOL(listIdx), sizeof(int), 0, hipMemcpyDeviceToHost));

    gpuErrchk( hipMemcpy(arr, gpu_coord, sizeof(Coordinate ) * count, hipMemcpyDeviceToHost) );

    return count;
}
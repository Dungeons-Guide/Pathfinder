#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../raycast.h"
#include "PathfindRequest.h"


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__device__ int listIdx;

__device__ bool ashadowcast(bool* req,
                          int lenX, int lenY, int lenZ,
                          float locX, float locY, float locZ,
                          float targetX, float  targetY, float  targetZ) {

    double dx = locX - targetX;
    double dy = locY - targetY;
    double dz = locZ - targetZ;
    short maxVal = max(abs(dx), max(abs(dy), abs(dz)));

    dx /= maxVal;
    dy /= maxVal;
    dz /= maxVal;

    for (short i = 0; i < maxVal; i++) {
        targetX += dx;
        targetY += dy;
        targetZ += dz;

        short x = (short) targetX;
        short y = (short) targetY;
        short z = (short) targetZ;

        int idx = z * lenX * lenY + y * lenX + x;
        if (req[idx]) {
            return false;
        }
    }
    return true;
}

__global__ void shadowcast(bool *req, int lenX, int lenY, int lenZ,
                           int offX, int offY, int offZ,
                           int resLenx, int resLeny, int resLenz,
                           short targetX, short targetY, short targetZ,
                           float offset, int rad,
                           Coordinate* arr) {
    short locX = blockIdx.x * blockDim.x + threadIdx.x + offX;
    short locY = blockIdx.y * blockDim.y + threadIdx.y + offY;
    short locZ = blockIdx.z * blockDim.z + threadIdx.z + offZ;

    short dx = targetX - locX;
    short dy = targetY - locY;
    short dz = targetZ - locZ;

    if (locX < offX || locY < offY || locZ < offZ || locX >= resLenx + offX || locY >= resLeny + offY || locZ >= resLenz + offZ) {
        return;
    }
    if (dx * dx + dy * dy + dz * dz > rad * rad) {
        return;
    }


    for (int z = 0; z < 2; z++) {
        for (int x = 0; x < 2; x++) {
            for (int y = 0; y < 2; y++) {
                int idx = (locZ-z) * lenX * lenY + (locY-2-y) * lenX + locX-x;
                if (idx < 0) continue;
                if (req[idx]) {
                    goto label;
                }
            }
        }
    }
    return;
    label:

    for (int x = 0; x < 1; x++) {
        for (int z = 0; z < 1; z++) {
            for (int y = 0; y < 2; y++) {
                bool flag = false;
                int xTarget;
                if (dx < 0) {
                    xTarget = targetX + 0.5 + offset;
                } else {
                    xTarget = targetX + 0.5 - offset;
                }
                flag |= ashadowcast(req, lenX, lenY, lenZ, locX + x/2.0, locY+ y / 2.0, locZ + z/2.0, xTarget, targetY + 0.5,
                                    targetZ + 0.5);

                if (dy < 0) {
                    xTarget = targetY + 0.5 + offset;
                } else {
                    xTarget = targetY + 0.5 - offset;
                }
                flag |= ashadowcast(req, lenX, lenY, lenZ, locX + x/2.0, locY + y / 2.0, locZ + z/2.0, targetX + 0.5, xTarget,
                                    targetZ + 0.5);
                
                if (dz < 0) {
                    xTarget = targetZ + 0.5 + offset;
                } else {
                    xTarget = targetZ + 0.5 - offset;
                }
                flag |= ashadowcast(req, lenX, lenY, lenZ, locX + x/2.0, locY + y / 2.0, locZ+ z/2.0, targetX + 0.5, targetY + 0.5,
                                   xTarget);

                if (flag) {
                    int val = atomicAdd(&listIdx, 1);
                    arr[val] = {
                            locX * 2 + x, locY * 2 + y, locZ * 2 + z
                    };
                }
            }

        }
    }

}

Coordinate* gpu_coord;
void setupCudaMemory() {
    gpuErrchk( hipMalloc((void**) &gpu_coord, sizeof(Coordinate) * GPU_RETURN_SIZE) );
}

int callShadowCast(bool *req, int lenX, int lenY, int lenZ,
                    int fromX, int fromY, int fromZ, int toX, int toY, int toZ,
                   short targetX, short targetY, short targetZ, float offset, int rad, Coordinate* arr) {

    int count = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(listIdx), &count, sizeof(int), 0, hipMemcpyHostToDevice);


    dim3 blockDim(32,1,32);
        dim3 gridDim(ceil((toX - fromX) / 32.0),ceil((toY - fromY) / 1.0),ceil((toZ - fromZ) / 32.0));
    shadowcast<<<gridDim, blockDim>>>(req, lenX, lenY, lenZ, fromX, fromY, fromZ,
                                      (toX - fromX), (toY - fromY), (toZ - fromZ),
                                      targetX, targetY, targetZ, offset, rad, gpu_coord);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    gpuErrchk(hipMemcpyFromSymbol(&count, HIP_SYMBOL(listIdx), sizeof(int), 0, hipMemcpyDeviceToHost));

    std::cout << count << std::endl;
    gpuErrchk( hipMemcpy(arr, gpu_coord, sizeof(Coordinate ) * count, hipMemcpyDeviceToHost) );

    return count;
}